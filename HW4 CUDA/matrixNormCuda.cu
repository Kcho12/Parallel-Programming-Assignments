/* Matrix normalization.
 compile with "nvcc matrixNormCuda.cu -o matrixNormCuda""
 */

 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>
 #include <time.h>
 #include <sys/time.h>
 #include <math.h>
 
 /* Program Parameters */
 #define MAXN 6000  /* Matrix size */
 int N = 6000;
 dim3 dG_numBlocks; //dG - dimension and size of grid in blocks
 dim3 dB_numThreads; //dB - dimension and size of blocks in threads
 
 /* Matrices */
 float A[MAXN * MAXN], B[MAXN * MAXN];
 float *A_d, *B_d; // device data equivalent to A and B
 
 /* Set the program parameters from the command-line arguments */
 void parameters(int argc, char **argv) {//added parameters from guass_serial.c so we can define the function dimensions
     int seed = 0;  /* Random seed */
     // char uid[32]; /*User name */
     dG_numBlocks = 32; //default values if we just want to quickly test matrixNorm with just ./matrixNorm
     dB_numThreads = 64;
 
     /* Read command-line arguments */
     srand((unsigned)time(NULL));  /* Randomize */
 
     if (argc == 5) {
         seed = atoi(argv[2]);
         srand(seed);
         dG_numBlocks=atoi(argv[3]);
         dB_numThreads=atoi(argv[4]);
         printf("Random seed = %i\n", seed);
         printf("Number of Blocks = %i\n", dG_numBlocks);
         printf("Number of Threads per block = %i\n", dB_numThreads);
     }
     if (argc == 4) { // ./matrixNormCuda <matrix_dimension> <number of Blocks> <number of threads per block>
        dG_numBlocks=atoi(argv[2]);
        dB_numThreads=atoi(argv[3]);
        printf("Number of Blocks = %i\n", dG_numBlocks);
        printf("Number of Threads per block = %i\n", dB_numThreads);
    }
     if (argc == 3) {// ./matrixNormCuda <matrix_dimension> [random seed] 
        seed = atoi(argv[2]);
        srand(seed);
        printf("Random seed = %i\n", seed);
    }
     if (argc >= 2) {
         N = atoi(argv[1]);
         if (N < 1 || N > MAXN) {
             printf("N = %i is out of range.\n", N);
             exit(0);
         }
     }
     if (argc >= 1) {
        //do ./matrixNormCuda 6000
	}
     else {
         printf("Usage: %s <matrix_dimension> [random seed] <number of Blocks> <number of threads per block>\n",
                 argv[0]);    
         exit(0);
     }
 }
     
 /* Initialize A and B*/
 void initialize_inputs() {
     int row, col;
     
     //srand((unsigned)time(NULL));
     for (row = 0; row < N; row++) {
         for (col = 0; col < N; col++) {
             A[row * N + col] = (float)rand() / 32768.0;//we need to copy this from host to the device
             B[row * N + col] = 0.0;
         }
     }
     
 }

/* Print input matrices */
void print_inputs() {
    int row, col;

    if (N < 100) {
        printf("\nA =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%5.2f%s", A[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}      

 //Display output for correctness comparison
 void print_B() {
     int row, col;
 
     if (N < 10) {
         printf("\nB = [");
         for (row = 0; row < N; row++) {
             for (col = 0; col < N; col++) {
                 printf("%5.2f%s", B[row * N + col], (col < N-1) ? ", " : "]\n");
             }
         }
     }
 }
 
 /* Kernel function */
 __global__ void matrixNorm(float *A_d, float *B_d, int N) {
    int row;
    float mu, sigma; // Mean and Standard Deviation
 
    int col = blockIdx.x * blockDim.x + threadIdx.x;//Treat col as the same as idx
     
     if (col < N) {//col is now incremented based on the blockIdx and threadIdx
         mu = 0.0;
         for (row=0; row < N; row++)
             mu += A_d[row * N + col]; //expression must have a pointer-to-object type. Since we are incrementing by every column automatically, we can just traverse the rows by multiplying by N.
         mu /= (float) N;
        __syncthreads();//wait for all cuda threads to get the mean
         
 
         sigma = 0.0;
         for (row=0; row < N; row++)
             sigma += powf(A_d[row * N + col] - mu, 2.0);
         sigma /= (float) N;
         __syncthreads();//wait for all cuda threads to get the stdeviation
 
         sigma = sqrt(sigma);
         for (row=0; row < N; row++) {
             if (sigma == 0.0)
                 B_d[row * N + col] = 0.0;
             else
                 B_d[row * N + col] = (A_d[row * N + col] - mu) / sigma;
         }
     }
 }
 
 
 int main(int argc, char **argv) {
     /* Timing variables */
     struct timeval start, stop;  /* Elapsed times using gettimeofday() */
     struct timezone tzdummy;
     unsigned long long runtime;
     float elapsed=0;/* Cuda timing elements*/
     hipEvent_t cstart, cstop;
     hipEventCreate(&cstart);
     hipEventCreate(&cstop);
 
     /* Process program parameters */ //Imported from guass_serial.c
     parameters(argc, argv);
     
     /* Initialize A and B */
     initialize_inputs();

     //print_inputs();
 
     /* Start Clock */
     printf("\n---------------------------------------------\n");
     printf("Matrix size N = %d", N);
     printf("\nStarting clock.\n\n");
     gettimeofday(&start, &tzdummy);
     hipEventRecord(cstart, 0);
     
    /*Data movement from Host to Device*/
     //cudaMalloc(void ** pointer, size_t nbytes)
     hipMalloc((void **) &A_d, sizeof(float)*N*N);//These mallocs have insane overhead. They add 1000ms to the runtime.
     hipMalloc((void **) &B_d, sizeof(float)*N*N);
 
     //cudaMemcpy(void *dst, void *src, size_t nbytes, enum cudaMemcpyKind direction);
     hipMemcpy(A_d, A, sizeof(float)*N*N, hipMemcpyHostToDevice); //copy A matrix from host to A_d matrix in devices
     
     /* Matrix Normalization */
     printf("Computing with Cuda.\n");
     matrixNorm<<<dG_numBlocks, dB_numThreads>>>(A_d, B_d, N);
     hipDeviceSynchronize(); //Fixes the timing for GPU. Makes sure all cuda threads are done before stoping the clock
   
     /* Stop Clock */
     gettimeofday(&stop, &tzdummy);
     runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);

    hipEventRecord(cstop, 0);
    hipEventSynchronize (cstop);

    hipEventElapsedTime(&elapsed, cstart, cstop);
 
     /* Display output */
     hipMemcpy(B, (B_d), sizeof(float)*N*N, hipMemcpyDeviceToHost); //send device B back to Host B for display
     print_B();
     
     hipFree(A_d);
     hipFree(B_d);
     
     /* Display timing results */
     printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
     printf("\nTime in gpu was %.2f ms\n", elapsed); //Time in GPU is more important for how efficient our algorithm is than total runtime.
     printf("\nStopped clock.");
     printf("\n---------------------------------------------\n");
     
     exit(0);
 }